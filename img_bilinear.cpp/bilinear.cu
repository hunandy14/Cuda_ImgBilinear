#include "hip/hip_runtime.h"
/***************************************************************************************
Name :
Date : 2018/01/08
By   : CharlotteHonG
Final: 2018/01/08
***************************************************************************************/
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <vector>
using namespace std;

#include "CudaMem\CudaMem.cuh"
#include "timer\timer.hpp"

#define BLOCK_DIM 16

__host__ __device__
inline static float bilinearRead(const float* img, 
	size_t width, float y, float x) // �u�ʨ���
{
	// ����F�I(����� 1+)
	size_t x0 = floor(x);
	size_t x1 = ceil(x);
	size_t y0 = floor(y);
	size_t y1 = ceil(y);
	// ������(�u��� 1-)
	float dx1 = x - x0;
	float dx2 = 1 - dx1;
	float dy1 = y - y0;
	float dy2 = 1 - dy1;
	// ����I
	const float& A = img[y0*width + x0];
	const float& B = img[y0*width + x1];
	const float& C = img[y1*width + x0];
	const float& D = img[y1*width + x1];
	// ���X���(�n��e)
	float AB = A*dx2 + B*dx1;
	float CD = C*dx2 + D*dx1;
	float X = AB*dy2 + CD*dy1;
	return X;
}
//======================================================================================
// �ŧiGPU���z�ܼ�(�u������)
texture<float, 2, hipReadModeElementType> rT;
// ���z�u�ʨ��Ȯ֤�
__global__ void biliner_texture_kernel(float* dst, int srcW, int srcH, float ratio) {
	int idxX = blockIdx.x * blockDim.x + threadIdx.x,
		idxY = blockIdx.y * blockDim.y + threadIdx.y;
	if(idxX < srcW*ratio && idxY < srcH*ratio) { // �|�h�]�@�I�I�n�ױ�
		float srcX = idxX / ratio;
		float srcY = idxY / ratio;
		size_t idx = (idxY*srcW*ratio + idxX);
		dst[idx] = tex2D(rT, srcX+0.5, srcY+0.5);
	}
}
// ���z�u�ʨ��Ȩ禡
#define AutoMem_Style
#ifdef AutoMem_Style
__host__ void biliner_texture_core(float *dst, const float* src,
	size_t dstW, size_t dstH, float ratio)
{
	Timer T; T.priSta = 1;
	// �]�mGPU�һݪ���
	int srcSize = dstW*dstH;
	int dstSize = srcSize*ratio*ratio;

	// �ŧi texture2D�}�C�øj�w
	T.start();
	CudaMemArr<float> cuArray(src, dstW, dstH);
	hipBindTextureToArray(rT, cuArray);
	T.print("  GPU new ���z�Ŷ�+�ƻs");

	// �]�m ���ӼҦ�and�W�X��ɸ����
	rT.filterMode = hipFilterModeLinear;
	rT.addressMode[0] = hipAddressModeClamp;
	rT.addressMode[1] = hipAddressModeClamp;

	// �n�DGPU�Ŷ�
	T.start();
	CudaData<float> gpu_dst(dstSize);
	T.print("  GPU new �@��Ŷ�");

	// �]�m�����
	dim3 block(BLOCK_DIM, BLOCK_DIM);
	dim3 grid(ceil((float)dstW*ratio / BLOCK_DIM), ceil((float)dstH*ratio / BLOCK_DIM));
	T.start();
	biliner_texture_kernel <<< grid, block >> > (gpu_dst, dstW, dstH, ratio);
	T.print("  �֤߭p��");

	// ���XGPU��
	T.start();
	gpu_dst.memcpyOut(dst, dstSize);
	T.print("  GPU ���X���");
}

#else
__host__ void biliner_texture_core(float *dst, const float* src,
	size_t dstW, size_t dstH, float ratio)
{
	Timer T; T.priSta = 1;
	// �]�mGPU�һݪ���
	int srcSize = dstW*dstH;
	int dstSize = srcSize*ratio*ratio;

	// �ŧi texture2D�}�C�øj�w
	T.start();
	hipChannelFormatDesc chDesc = hipCreateChannelDesc<float>();
	hipArray* cuArray = nullptr;
	hipMallocArray(&cuArray, &chDesc, dstW, dstH);
	hipMemcpyToArray(cuArray, 0, 0, src, srcSize*sizeof(float), hipMemcpyHostToDevice);
	hipBindTextureToArray(rT, cuArray);
	T.print("  GPU new ���z�Ŷ�");

	// �]�m ���ӼҦ�and�W�X��ɸ����
	rT.filterMode = hipFilterModeLinear;
	rT.addressMode[0] = hipAddressModeClamp;
	rT.addressMode[1] = hipAddressModeClamp;

	// �n�DGPU�Ŷ�
	T.start();
	float* gpu_dst = nullptr;
	hipMalloc((void**)&gpu_dst, dstSize*sizeof(float));
	T.print("  GPU new �@��Ŷ�");

	// �]�m�����
	dim3 block(BLOCK_DIM, BLOCK_DIM);
	dim3 grid(ceil((float)dstW*ratio / BLOCK_DIM), ceil((float)dstH*ratio / BLOCK_DIM));
	T.start();
	biliner_kernel << < grid, block >> > (gpu_dst, dstW, dstH, ratio);
	T.print("  �֤߭p��");

	// ���XGPU��
	T.start();
	hipMemcpy(dst, gpu_dst, dstSize*sizeof(float), hipMemcpyDeviceToHost);
	T.print("  GPU ���X���");

	// ����GPU�O����
	hipUnbindTexture(rT);
	hipFreeArray(cuArray);
	hipFree(gpu_dst);
}
#endif // AutoMem_Style

// ���z�u�ʨ��Ȩ禡 vector �श����
__host__ double biliner_texture(vector<float>& dst, const vector<float>& src,
	size_t width, size_t height, float ratio)
{
	Timer T; T.priSta = 1;
	T.start();
	dst.resize(width*ratio * height*ratio);
	T.print(" CPU new �x�s�Ŷ�");
	T.start();
	biliner_texture_core(dst.data(), src.data(), width, height, ratio);
	T.print(" GPU ����");
	return T;
}



//======================================================================================
// �@�ɰO����u�ʨ��Ȯ֤�
__global__ void biliner_share_kernel(float* dst, const float* src, int srcW, int srcH, float ratio) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int newH = (int)(floor(srcH * ratio));
	int newW = (int)(floor(srcW * ratio));
	if(i < srcW*ratio && j < srcH*ratio) { // �|�h�]�@�I�I�n�ױ�
		// �վ���
		float srcY, srcX;
		if (ratio < 1) {
			srcY = ((j+0.5f)/ratio) - 0.5;
			srcX = ((i+0.5f)/ratio) - 0.5;
		} else {
			srcY = j * (srcH-1.f) / (newH-1.f);
			srcX = i * (srcW -1.f) / (newW-1.f);
		}
		// ������ɭ�
		dst[j*newW + i] = bilinearRead(src, srcW, srcY, srcX);
	}
}
// �@�ɰO����u�ʨ��Ȩ禡
__host__ void biliner_share_core(float *dst, const float* src,
	size_t srcW, size_t srcH, float ratio)
{
	Timer T; T.priSta = 1;
	// �]�mGPU�һݪ���
	int srcSize = srcW*srcH;
	int dstSize = srcSize*ratio*ratio;

	// �n�DGPU�Ŷ�
	T.start();
	CudaData<float> gpu_src(srcSize);
	T.print("  GPU new �Ŷ�1");
	T.start();
	CudaData<float> gpu_dst(dstSize);
	T.print("  GPU new �Ŷ�2");
	// �ƻs��GPU
	T.start();
	gpu_src.memcpyIn(src, srcSize);
	T.print("  GPU �ƻs");

	// �]�m�����
	dim3 block(BLOCK_DIM, BLOCK_DIM);
	dim3 grid(ceil((float)srcW*ratio / BLOCK_DIM), ceil((float)srcH*ratio / BLOCK_DIM));
	T.start();
	biliner_share_kernel <<< grid, block >> > (gpu_dst, gpu_src, srcW, srcH, ratio);
	T.print("  �֤߭p��");

	// ���XGPU��
	T.start();
	gpu_dst.memcpyOut(dst, dstSize);
	T.print("  GPU ���X���");

	// ����GPU�Ŷ�
	T.start();
	gpu_src.~CudaData();
	gpu_dst.~CudaData();
	T.print("  GPU ����Ŷ�");
}

// �@�ɰO����u�ʨ��Ȩ禡 vector �श����
__host__ double biliner_share(vector<float>& dst, const vector<float>& src,
	size_t width, size_t height, float ratio)
{
	Timer T; T.priSta = 1;
	T.start();
	dst.resize(width*ratio * height*ratio);
	T.print(" CPU new �x�s�Ŷ�");
	T.start();
	biliner_share_core(dst.data(), src.data(), width, height, ratio);
	T.print(" GPU ����");
	return T;
}



//======================================================================================
__host__ void biliner_CPU_core(vector<float>& img, const vector<float>& img_ori, 
	size_t width, size_t height, float Ratio)
{
	int newH = static_cast<int>(floor(height * Ratio));
	int newW = static_cast<int>(floor(width  * Ratio));
	img.resize(newH*newW);
	// �]�s�Ϯy��
	for (int j = 0; j < newH; ++j) {
		for (int i = 0; i < newW; ++i) {
			// �վ���
			float srcY, srcX;
			if (Ratio < 1) {
				srcY = ((j+0.5f)/Ratio) - 0.5;
				srcX = ((i+0.5f)/Ratio) - 0.5;
			} else {
				srcY = j * (height-1.f) / (newH-1.f);
				srcX = i * (width -1.f) / (newW-1.f);
			}
			// ������ɭ�
			img[j*newW + i] = bilinearRead(img_ori.data(), width, srcY, srcX);
		}
	}
}
__host__ double biliner_CPU(vector<float>& dst, const vector<float>& src,
	size_t width, size_t height, float ratio)
{
	Timer T; T.priSta = 1;
	T.start();
	dst.resize(width*ratio * height*ratio);
	T.print(" CPU new �x�s�Ŷ�");
	T.start();
	biliner_CPU_core(dst, src, width, height, ratio);
	T.print(" CPU ����");
	return T;
}
//======================================================================================


