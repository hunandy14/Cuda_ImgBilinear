#include "hip/hip_runtime.h"
/*****************************************************************
Name :
Date : 2018/01/08
By   : CharlotteHonG
Final: 2018/01/08
*****************************************************************/
#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <vector>
using namespace std;

#include "CudaMem\CudaMem.cuh"
#include "timer.hpp"

#define BLOCK_DIM 16

// �ŧiGPU���z�ܼ�(�u������)
texture<float, 2, hipReadModeElementType> rT;
// ���z�u�ʨ��Ȯ֤�
__global__ void biliner_kernel(float* dst, int srcW, int srcH, float ratio) {
	int idxX = blockIdx.x * blockDim.x + threadIdx.x,
		idxY = blockIdx.y * blockDim.y + threadIdx.y;
	if(idxX < srcW*ratio && idxY < srcH*ratio) { // �|�h�]�@�I�I�n�ױ�
		float srcX = idxX / ratio;
		float srcY = idxY / ratio;
		size_t idx = (idxY*srcW*ratio + idxX);
		dst[idx] = tex2D(rT, srcX+0.5, srcY+0.5);
	}
}
// ���z�u�ʨ��Ȩ禡
#define AutoMem_Style
#ifdef AutoMem_Style
__host__ void biliner_texture_core(float *dst, const float* src,
	size_t dstW, size_t dstH, float ratio)
{
	Timer T; T.priSta = 1;
	// �]�mGPU�һݪ���
	int srcSize = dstW*dstH;
	int dstSize = srcSize*ratio*ratio;

	// �ŧi texture2D�}�C�øj�w
	T.start();
	CudaMemArr<float> cuArray(src, dstW, dstH);
	hipBindTextureToArray(rT, cuArray);
	T.print("  GPU new ���z�Ŷ�");

	// �]�m ���ӼҦ�and�W�X��ɸ����
	rT.filterMode = hipFilterModeLinear;
	rT.addressMode[0] = hipAddressModeClamp;
	rT.addressMode[1] = hipAddressModeClamp;

	// �n�DGPU�Ŷ�
	T.start();
	CudaData<float> gpu_dst(dstSize);
	T.print("  GPU new �@��Ŷ�");

	// �]�m�����
	dim3 block(BLOCK_DIM, BLOCK_DIM);
	dim3 grid(ceil((float)dstW*ratio / BLOCK_DIM), ceil((float)dstH*ratio / BLOCK_DIM));
	T.start();
	biliner_kernel <<< grid, block >> > (gpu_dst, dstW, dstH, ratio);
	T.print("  �֤߭p��");

	// ���XGPU��
	T.start();
	gpu_dst.memcpyOut(dst, dstSize);
	T.print("  GPU ���X���");
}

#else
__host__ void biliner_texture_core(float *dst, const float* src,
	size_t dstW, size_t dstH, float ratio)
{
	Timer T; T.priSta = 1;
	// �]�mGPU�һݪ���
	int srcSize = dstW*dstH;
	int dstSize = srcSize*ratio*ratio;

	// �ŧi texture2D�}�C�øj�w
	T.start();
	hipChannelFormatDesc chDesc = hipCreateChannelDesc<float>();
	hipArray* cuArray = nullptr;
	hipMallocArray(&cuArray, &chDesc, dstW, dstH);
	hipMemcpyToArray(cuArray, 0, 0, src, srcSize*sizeof(float), hipMemcpyHostToDevice);
	hipBindTextureToArray(rT, cuArray);
	T.print("  GPU new ���z�Ŷ�");

	// �]�m ���ӼҦ�and�W�X��ɸ����
	rT.filterMode = hipFilterModeLinear;
	rT.addressMode[0] = hipAddressModeClamp;
	rT.addressMode[1] = hipAddressModeClamp;

	// �n�DGPU�Ŷ�
	T.start();
	float* gpu_dst = nullptr;
	hipMalloc((void**)&gpu_dst, dstSize*sizeof(float));
	T.print("  GPU new �@��Ŷ�");

	// �]�m�����
	dim3 block(BLOCK_DIM, BLOCK_DIM);
	dim3 grid(ceil((float)dstW*ratio / BLOCK_DIM), ceil((float)dstH*ratio / BLOCK_DIM));
	T.start();
	biliner_kernel << < grid, block >> > (gpu_dst, dstW, dstH, ratio);
	T.print("  �֤߭p��");

	// ���XGPU��
	T.start();
	hipMemcpy(dst, gpu_dst, dstSize*sizeof(float), hipMemcpyDeviceToHost);
	T.print("  GPU ���X���");

	// ����GPU�O����
	hipUnbindTexture(rT);
	hipFreeArray(cuArray);
	hipFree(gpu_dst);
}
#endif // AutoMem_Style

// ���z�u�ʨ��Ȩ禡 vector �श����
__host__ void biliner_texture(vector<float>& dst, const vector<float>& src,
	size_t width, size_t height, float ratio)
{
	Timer T; T.priSta = 0;
	T.start();
	dst.resize(width*ratio * height*ratio);
	T.print(" CPU new �x�s�Ŷ�");
	T.start();
	biliner_texture_core(dst.data(), src.data(), width, height, ratio);
	T.print(" GPU ����");
}











